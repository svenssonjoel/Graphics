#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h> 


#define WIDTH 512
#define HEIGHT 512
#define ITERS 512

#define N (WIDTH*HEIGHT)

#define max_size   4
#define max_colors 16

#define xmax  1.2f
#define xmin -2.0f
#define ymax  1.2f
#define ymin -1.2f

#define deltaP ((xmax - xmin)/512)
#define deltaQ ((ymax - ymin)/512) 


extern "C" __global__ void mandelGen(uint8_t* output0)
{
    extern __shared__ uint8_t sbase[];
    uint32_t tid = threadIdx.x;
    float v0;
    float v1;
    uint32_t v2;
    float t3;
    float t4;
    uint32_t t5;
    
    if (blockIdx.x < 512U) {
        v0 = 0.0F;
        v1 = 0.0F;
        v2 = 1U;
        while (v0 * v0 + v1 * v1 < 4.0F && v2 < 512U) {
            t3 = v0;
            t4 = v1;
            t5 = v2;
            v0 = t3 * t3 - t4 * t4 + (-0.7931140065193176F + (float) tid *
                                      1.3693165965378284e-4F);
            v1 = 2.0F * t3 * t4 + (0.1409740000963211F -
                                   (float) blockIdx.x *
                                   2.0146874885540456e-4F);
            v2 = t5 + 1U;
        }
        output0[blockIdx.x * 512U + tid] = (uint8_t) v2 % 16U * 16U;
    }
}


__global__ void kernel(uint8_t* output0){
  
    float v3;
    float v2;
    uint32_t v1;
    v3 = 0.0f;
    v2 = 0.0f;
    v1 = 1;
    while (((((v3*v3)+(v2*v2))<4.0f)&&(v1<512))){
      
        float t6;
        float t5;
        uint32_t t4;
        t6 = v3;
        t5 = v2;
        t4 = v1;
        v3 = (((t6*t6)-(t5*t5))+(-2.0f+(((float)threadIdx.x)*6.25e-3f)));
        v2 = (((2.0f*t6)*t5)+(1.2f-(((float)blockIdx.x)*4.6875e-3f)));
        v1 = (t4+1);
        
    }
    output0[((blockIdx.x*512)+threadIdx.x)] = ((((uint8_t)v1)%16)*16);
    
  
}

__global__ void plate1(uint8_t* output0){
  
    float v3;
    float v2;
    uint32_t v1;
    v3 = 0.0f;
    v2 = 0.0f;
    v1 = 1;
    while (((((v3*v3)+(v2*v2))<4.0f)&&(v1<512))){
      
        float t6;
        float t5;
        uint32_t t4;
        t6 = v3;
        t5 = v2;
        t4 = v1;
        v3 = (((t6*t6)-(t5*t5))+(-0.69106f+(((float)threadIdx.x)*3.008172e-7f)));
        v2 = (((2.0f*t6)*t5)+(0.387228f-(((float)blockIdx.x)*2.4418114e-7f)));
        v1 = (t4+1);
        
    }
    output0[((blockIdx.x*512)+threadIdx.x)] = ((((uint8_t)v1)%16)*16);
    
  
}

__global__ void plate2(uint8_t* output0){
  
    float v3;
    float v2;
    uint32_t v1;
    v3 = 0.0f;
    v2 = 0.0f;
    v1 = 1;
    while (((((v3*v3)+(v2*v2))<4.0f)&&(v1<512))){
      
        float t6;
        float t5;
        uint32_t t4;
        t6 = v3;
        t5 = v2;
        t4 = v1;
        v3 = (((t6*t6)-(t5*t5))+(-0.793114f+(((float)threadIdx.x)*1.3693166e-4f)));
        v2 = (((2.0f*t6)*t5)+(0.140974f-(((float)blockIdx.x)*2.0146875e-4f)));
        v1 = (t4+1);
        
    }
    output0[((blockIdx.x*512)+threadIdx.x)] = ((((uint8_t)v1)%16)*16);
    
  
}
__global__ void plate3(uint8_t* output0){
  
    float v3;
    float v2;
    uint32_t v1;
    v3 = 0.0f;
    v2 = 0.0f;
    v1 = 1;
    while (((((v3*v3)+(v2*v2))<4.0f)&&(v1<512))){
      
        float t6;
        float t5;
        uint32_t t4;
        t6 = v3;
        t5 = v2;
        t4 = v1;
        v3 = (((t6*t6)-(t5*t5))+(-0.745464f+(((float)threadIdx.x)*1.4854595e-7f)));
        v2 = (((2.0f*t6)*t5)+(0.11303f-(((float)blockIdx.x)*1.23051e-7f)));
        v1 = (t4+1);
        
    }
    output0[((blockIdx.x*512)+threadIdx.x)] = ((((uint8_t)v1)%16)*16);
    
  
}


__global__ void mandel(uint8_t *out) { 
  
  int bid = blockIdx.x; 
  int tid = threadIdx.x; 

  float x = 0.0, y = 0.0, xsq = 0.0, ysq = 0.0;
  int color = 1; 

  while (color < ITERS && (xsq + ysq) < max_size) {

    xsq = x*x;
    ysq = y*y;
    y = 2*x*y+(ymax - blockIdx.x*deltaQ);
    x = xsq - ysq + (xmin + threadIdx.x * deltaP);
    color ++;
  }
  
  
  out[bid* 512 + tid] = (color % 8) * 32; // % max_colors;

  } 

/* ------------------------------------------------------------------------
   MAIN
   --------------------------------------------------------------------- */
int main(void) {
  
   
  uint8_t *r; 

  uint8_t *dr; 
  
  r = (uint8_t*)malloc(sizeof(uint8_t) * N);
  
  hipMalloc((void**)&dr,N*sizeof(uint8_t));

  hipMemset(dr,0,N*sizeof(uint8_t));
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  //mandel<<<HEIGHT,WIDTH,0>>>(dr);
  //kernel<<<HEIGHT,WIDTH,0>>>(dr);	
  //plate1<<<HEIGHT,WIDTH,0>>>(dr);	
  //plate2<<<HEIGHT,WIDTH,0>>>(dr);	  		
  //plate3<<<HEIGHT,WIDTH,0>>>(dr);	  		

  mandelGen<<<HEIGHT,WIDTH,0>>>(dr);	  		

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  // std::cout << std::endl;

  hipMemcpy(r,dr,N*sizeof(uint8_t),hipMemcpyDeviceToHost);
  //cudaMemcpy(m,dm,64*sizeof(float),cudaMemcpyDeviceToHost);


  for (int i = 0; i < N; i ++) { 
    printf("%d ",r[i]);
  }

    
  printf("Elapsed time: %f\n", elapsedTime);


  FILE *file; 
  file = fopen("image.out","w");
  fwrite(r,sizeof(uint8_t),N,file);
  fclose(file);
  

  return 0;
}


