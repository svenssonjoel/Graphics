#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h> 


// For comparisons
//#include "seqScan.c"

__device__ int sklansky(int i, float* input0, float *output0, uint8_t *sbase,float *maxs) {

    uint32_t t2 = ((blockIdx.x*32)+((threadIdx.x&4294967294)|(threadIdx.x&1)));
    uint32_t t9 = ((threadIdx.x&4294967292)|(threadIdx.x&3));
    uint32_t t14 = ((threadIdx.x&4294967288)|(threadIdx.x&7));
    uint32_t t19 = ((threadIdx.x&4294967280)|(threadIdx.x&15));
    ((float*)sbase)[threadIdx.x] = (((threadIdx.x&1)<1) ? input0[t2] : (input0[((blockIdx.x*32)+((threadIdx.x&4294967294)|0))]+input0[t2]));
    //__syncthreads();
    ((float*)(sbase+128))[threadIdx.x] = (((threadIdx.x&3)<2) ? ((float*)sbase)[t9] : (((float*)sbase)[((threadIdx.x&4294967292)|1)]+((float*)sbase)[t9]));
    //__syncthreads();
    ((float*)sbase)[threadIdx.x] = (((threadIdx.x&7)<4) ? ((float*)(sbase+128))[t14] : (((float*)(sbase+128))[((threadIdx.x&4294967288)|3)]+((float*)(sbase+128))[t14]));
    //__syncthreads();
    ((float*)(sbase+128))[threadIdx.x] = (((threadIdx.x&15)<8) ? ((float*)sbase)[t19] : (((float*)sbase)[((threadIdx.x&4294967280)|7)]+((float*)sbase)[t19]));
    //__syncthreads();
    ((float*)sbase)[threadIdx.x] = ((threadIdx.x<16) ? ((float*)(sbase+128))[threadIdx.x] : (((float*)(sbase+128))[15]+((float*)(sbase+128))[threadIdx.x]));
    //__syncthreads();
    
    output0[((blockIdx.x*32)+threadIdx.x)] = ((float*)sbase)[threadIdx.x];
 
    if (threadIdx.x == 0) 
      maxs[i] = ((float*)sbase)[31];
    
    return 0;
}

__global__ void kernel(float* input0,
                       float* output0, 
                       float* maxout){
   
  extern __shared__ __attribute__ ((aligned(16))) uint8_t sbase[];
  
  float *maxs = (float*)(sbase+(sizeof(float)*64));
  
  for (int i = 0; i < 32; i ++) { 

    sklansky(i,input0+i*32,output0+i*32,sbase,maxs);
  }
  
  float v; //  discard this value
  sklansky(0,maxs,maxs,sbase,&v);
  

  // distribute 
  if (threadIdx.x > 0) {
    for (int j = 0; j < 32; j ++) {
      output0[threadIdx.x*32+j] += maxs[threadIdx.x-1];
      
    }
  }
  
  maxout[threadIdx.x] = maxs[threadIdx.x];

}

#define N 32*32

int main(void) {
  
  float v[N]; 
  float r[N]; 
  //float rc[N];
  float m[32];

  float *dv; 
  float *dr; 
  float *dm;
  
  for (int i = 0; i < N; i ++) {
    v[i] = 1.0; 
    r[i] = 7.0;
  }

  hipMalloc((void**)&dv,N*sizeof(float)); 
  hipMalloc((void**)&dr,N*sizeof(float));
  hipMalloc((void**)&dm,32*sizeof(float));
  
  hipMemcpy(dv,v,N*sizeof(float),hipMemcpyHostToDevice);
  
  kernel<<<1,32,32*3*(sizeof(float))>>>(dv,dr,dm);

  hipMemcpy(r,dr,N*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(m,dm,32*sizeof(float),hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i ++) { 
    printf("%f ",r[i]);
  }

  
  printf("\n ------ \n");
  
  for (int i = 0; i < 32; i ++) { 
    printf("%f ",m[i]);
  }
  
  

  //seqScan(v,rc,N);
  //int s = compare(rc,r,0.01,N);
 
 

  //printf ("\n%s\n", s? "same" : "not the same");
  

  return 0;
}
