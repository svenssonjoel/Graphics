#include <hip/hip_runtime.h>
#include <stdio.h>



__global__ void scan_local(float *in, float *out) {
  out[0] = in[0];
  for (int i = 0; i < 32; i ++) {
    
    out[i] = out[i-1] + in[i]; 
  }

    
} 




int main(void) {
  
  float v[32]; 
  float r[32]; 

  float *dv; 
  float *dr; 

  for (int i = 0; i < 32; i ++) {
    v[i] = 1.0; 
    r[i] = 7.0;
  }

  hipMalloc((void**)&dv,32*sizeof(float)); 
  hipMalloc((void**)&dr,32*sizeof(float));
  
  hipMemcpy(dv,v,32*sizeof(float),hipMemcpyHostToDevice);
  
  scan_local<<<1,1,0>>>(dv,dr);

  hipMemcpy(r,dr,32*sizeof(float),hipMemcpyDeviceToHost);

  for (int i = 0; i < 32; i ++) { 
    printf("%f ",r[i]);
  }

  return 0;
}
