#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h> 


// For comparisons
//#include "seqScan.c"


#define CLONES 8
#define REPS   8

// block size in ELEMENTS!
#define BLOCK_SIZE (CLONES*REPS*64) 

#define N 4096*CLONES*REPS*64


/* ------------------------------------------------------------------------
   Unrolled in-place(shared memory) Scan without syncs (32 threads, 64 elts). 
   Needs 2*64 elements of shared memory storage (512bytes).
     (shared mem is 49152 bytes, but you share it with other blocks on an MP) 
   --------------------------------------------------------------------- */
__device__ void skl_scan(int i, 
			 float* input, 
			 float *output, 
			 float *s_data, // The shared memory
			 float *maxs) { 
    
  int tid = threadIdx.x; 
  int tids = tid << 1; 

  // Load data from global memory into shared memory (in two separate load ops)
  s_data[tid*2] = input[tid*2]; 
  s_data[tid*2+1] = input[tid*2+1]; 
  //  __syncthreads();


  s_data[tids | 1] += s_data[tids]; 
  s_data[(tids | 3) - (tid & 1)] += s_data[tids & 0xFFFFFFFC | 1]; 
  s_data[(tids | 7) - (tid & 3)] += s_data[tids & 0xFFFFFFF8 | 3]; 
  s_data[(tids | 15) - (tid & 7)] += s_data[tids & 0xFFFFFFF0 | 7]; 
  s_data[(tids | 31) - (tid & 15)] += s_data[tids & 0xFFFFFFE0 | 15]; 
  s_data[(tids | 63) - (tid & 31)] += s_data[tids & 0xFFFFFFC0 | 31];
  // NO Interleaved SYNCS here.

  //__syncthreads();
  output[tid*2] = s_data[tid*2]; 
  output[tid*2+1] = s_data[tid*2+1];
  
  //__syncthreads();
  if(tid % 32 == 0) 
    maxs[(i<<3)+(tid>>5)] = s_data[(tid << 1) | 0x3F];
  //maxs[i*CLONES+(tid / 32)] = s_data[(tid / 32)*64 + 63];
  // (i<<3)+(tid>>5)                    ((tid>>5)<<6) + 63 
  //                                    (tid << 1) | 0x3F)   
  
}


/* ------------------------------------------------------------------------
   The Scan kernel (Thousand(s) of elements. NO SYNCS AT ALL) 
   --------------------------------------------------------------------- */
__global__ void kernel(float* input0,
                       float* output0, 
                       float* maxout){
   
  // shared data. (two different kinds. warp local and across warps.) 
  extern __shared__ float s_data[]; 
  float *maxs = &s_data[512]; 
  
  // Sequentially execute 64 scans
  for (int i = 0; i < REPS; i ++) {  
    skl_scan(i,
	     input0+(blockIdx.x*BLOCK_SIZE)+(i*512),
	     output0+(blockIdx.x*BLOCK_SIZE)+(i*512),
	     s_data,maxs);
  }

  // Now needs one __syncthreads() here! 
  __syncthreads();

  // in parallel scan the maximum array 
  float v; //discard this value.
  if (threadIdx.x < 32) 
    skl_scan(0,maxs,maxs,(float *)s_data,&v);
  
  __syncthreads();

  
  // really messy code  
  for (int j = 0; j < REPS; j ++) {
    if (j != 0 || threadIdx.x >=  64) 
      output0[(blockIdx.x*BLOCK_SIZE)+(j*256)+threadIdx.x] += maxs[(((j*256)+threadIdx.x) / 64)-1];
    output0[(blockIdx.x*BLOCK_SIZE)+(j*256)+threadIdx.x+2048] += maxs[(((j*256)+threadIdx.x+2048) /64)-1];
  }
 
  // This is a debug step. 
  if (threadIdx.x < 32) {
    maxout[threadIdx.x] = maxs[threadIdx.x];
    maxout[threadIdx.x+32] = maxs[threadIdx.x+32];
  }
}

/* ------------------------------------------------------------------------
   MAIN
   --------------------------------------------------------------------- */
int main(void) {
  
  float *v; 
  float *r; 
  //float rc[N];
  float m[64];

  float *dv; 
  float *dr; 
  float *dm;
  
  v = (float*)malloc(sizeof(float) * N);
  r = (float*)malloc(sizeof(float) * N);
  memset(m,0,64*sizeof(float));
  
  for (int i = 0; i < N; i ++) {
    v[i] = 1.0; 
    r[i] = 7.0;
  }

  hipMalloc((void**)&dv,N*sizeof(float)); 
  hipMalloc((void**)&dr,N*sizeof(float));
  hipMalloc((void**)&dm,64*sizeof(float));
  
  hipMemcpy(dv,v,N*sizeof(float),hipMemcpyHostToDevice);
  
  //kernel<<<1,32,32*3*(sizeof(float))>>>(dv,dr,dm);
  //kernel<<<1,16,32*2*(sizeof(float))>>>(dv,dr,dm);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  kernel<<<4096,256,(512+64)*(sizeof(float))>>>(dv,dr,dm);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  // std::cout << std::endl;

  hipMemcpy(r,dr,N*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(m,dm,64*sizeof(float),hipMemcpyDeviceToHost);


  for (int i = 0; i < 4096 /*N*/; i ++) { 
    printf("%f ",r[i]);
  }

  
  printf("\n ------ \n");
  
  for (int i = 0; i < 64; i ++) { 
    printf("%f ",m[i]);
  }
  
  

  printf("Elapsed time: %f\n", elapsedTime);


  //seqScan(v,rc,N);
  //int s = compare(rc,r,0.01,N);
 
 

  //printf ("\n%s\n", s? "same" : "not the same");
  

  return 0;
}


